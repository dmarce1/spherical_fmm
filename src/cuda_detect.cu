/*****************
 *
 * THIS (slightly modifed) CODE TAKEN FROM
 * https://wagonhelm.github.io/articles/2018-03/detecting-cuda-capability-with-cmake
 *
 */



#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv){
    hipDeviceProp_t dP;
    float min_cc = 3.0;

    int rc = hipGetDeviceProperties(&dP, 0);
    if(rc != hipSuccess) {
        hipError_t error = hipGetLastError();
        printf("CUDA error: %s", hipGetErrorString(error));
        return rc; /* Failure */
    }
  /*  if( dP.major >= 8 ) {
    	dP.major = 7;
    	dP.minor = 5;
    }*/
    if((dP.major+(dP.minor/10)) < min_cc) {
        printf("Min Compute Capability of %2.1f required:  %d.%d found\n Not Building CUDA Code", min_cc, dP.major, dP.minor);
        return 1; /* Failure */
    } else {
        printf("%d%d", dP.major, dP.minor);
        return 0; /* Success */
    }
}

